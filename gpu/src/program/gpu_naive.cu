#include "hip/hip_runtime.h"
#include "move.h"
#include "read.h"
#include "utils.h"
#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#include <time.h>

// Function to multiply CSR matrix by a vector
__global__ void multiplyMatrixVector(float *matrix, int rows, int cols, float *vector, float *result) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= rows)
        return;

    float sum = 0.0;
    int row_start = i * cols;
    for (int j = 0; j < cols; j++) {
        sum += matrix[row_start + j] * vector[j];
    }

    result[i] = sum;
}

int main() {
    printf("Running!\n");

    // char filename[] = "../matrices/1138_bus_sorted.mtx";
    // char filename[] = "../matrices/4884_bcsstk16_sorted.mtx";
    char filename[] = "../matrices/10974_bcsstk17_sorted.mtx";
    // char filename[] = "../matrices/36057_onetone1_sorted.mtx";
    // char filename[] = "../matrices/929901_Hardesty2_sorted.mtx";
    // char filename[] = "../matrices/923136_Emilia_923_sorted.mtx";
    // Read the matrix from file
    int rows, cols, nnz;
    float *h_matrix = readMTXFile(filename, &rows, &cols, &nnz);
    float *d_matrix = moveArrayToDevice(h_matrix, cols * rows);

    // Create vector of ones
    float *h_vector = (float *)std::malloc(cols * sizeof(float));
    if (h_vector == NULL) {
        fprintf(stderr, "Memory allocation failed for vector\n");
        std::free(h_matrix);
        hipFree(d_matrix);
        exit(1);
    }

    for (int i = 0; i < cols; i++) {
        h_vector[i] = 1.0;
    }

    float *d_vector = moveArrayToDevice(h_vector, cols);

    // Multiply matrix by vector
    size_t results_size = rows * sizeof(float);
    float *h_results = (float *)std::malloc(results_size);
    float *d_results;
    hipMalloc(&d_results, results_size);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int threads_per_block = 256;
    int num_blocks = (rows + threads_per_block - 1) / threads_per_block;
    printf("Launching %d blocks of %d threads\n", num_blocks, threads_per_block);
    hipEventRecord(start);
    multiplyMatrixVector<<<num_blocks, 256>>>(d_matrix, rows, cols, d_vector, d_results);
    hipEventRecord(stop);
    hipError_t cudaerr = hipDeviceSynchronize();
    if (cudaerr != hipSuccess)
        printf("kernel launch failed with error \"%s\".\n", hipGetErrorString(cudaerr));
    hipEventSynchronize(stop);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
    }
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Save results
    hipMemcpy(h_results, d_results, results_size, hipMemcpyDeviceToHost);
    char result_path[256];
    sprintf(result_path, "../results/%s_result.txt", strrchr(filename, '/') ? strrchr(filename, '/') + 1 : filename);
    writeVectorToFile(result_path, h_results, rows);

    printf("Kernel Time: %f ms\n", milliseconds);

    // Free allocated memory
    hipFree(d_vector);
    hipFree(d_results);
    hipFree(d_matrix);
    std::free(h_vector);
    std::free(h_results);
    std::free(h_matrix);

    return 0;
}
