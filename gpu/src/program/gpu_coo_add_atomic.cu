#include "hip/hip_runtime.h"
#include "move.h"
#include "read.h"
#include "readcu.h"
#include "timers.h"
#include "utils.h"
#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#include <time.h>

#define WARMUP 3
#define NITER 10
// Function to multiply CSR matrix by a vector
__global__ void multiplyMatrixVector(COOMatrix matrix, float *vector, float *result) {

    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= matrix.nnz)
        return;

    int row = matrix.rows[i];
    int col = matrix.cols[i];
    float val = matrix.values[i];

    atomicAdd(&result[row], val * vector[col]);
}

int main() {
    printf("Running!\n");

    // char filename[] = "../matrices/1138_bus_sorted.mtx";
    // char filename[] = "../matrices/4884_bcsstk16_sorted.mtx";
    // char filename[] = "../matrices/10974_bcsstk17_sorted.mtx";
    // char filename[] = "../matrices/36057_onetone1_sorted.mtx";
    // char filename[] = "../matrices/929901_Hardesty2_sorted.mtx";
    // char filename[] = "../matrices/923136_Emilia_923_sorted.mtx";

    auto filenames = getFilenames();
    for (std::string filename : filenames) {
        // Read the matrix from file
        COOMatrix h_matrix = readMTXFileCOO(filename.c_str());
        COOMatrix d_matrix = h_matrix;
        moveCOOToDevice(h_matrix, d_matrix);

        int cols = h_matrix.numCols, rows = h_matrix.numRows;

        // Create vector of ones
        float *h_vector = (float *)std::malloc(cols * sizeof(float));
        if (h_vector == NULL) {
            fprintf(stderr, "Memory allocation failed for vector\n");
            freeCOOMatrixCuda(h_matrix);
            exit(1);
        }

        for (int i = 0; i < cols; i++) {
            h_vector[i] = 1.0;
        }

        float *d_vector = moveArrayToDevice(h_vector, cols);

        // Multiply matrix by vector
        size_t results_size = rows * sizeof(float);
        float *h_results = (float *)std::malloc(results_size);
        float *d_results;
        hipMalloc(&d_results, results_size);

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        int threads_per_block = 256;
        int num_blocks = (d_matrix.nnz + threads_per_block - 1) / threads_per_block;
        printf("Launching %d blocks of %d threads\n", num_blocks, threads_per_block);
        float timers[NITER];
        float iter_time = 0;
        for (int i = -WARMUP; i < NITER; i++) {
            hipMemset(d_results, 0, results_size);
            hipEventRecord(start);
            multiplyMatrixVector<<<num_blocks, threads_per_block>>>(d_matrix, d_vector, d_results);
            hipEventRecord(stop);

            hipError_t cudaerr = hipDeviceSynchronize();
            if (cudaerr != hipSuccess)
                printf("kernel launch failed with error \"%s\".\n", hipGetErrorString(cudaerr));

            hipEventSynchronize(stop);
            hipEventElapsedTime(&iter_time, start, stop);
            if (i >= 0) {
                timers[i] = iter_time;
                printf("%d iter_time %f\n", i, iter_time);
            }
        }

        // Save results
        hipMemcpy(h_results, d_results, results_size, hipMemcpyDeviceToHost);
        char result_path[256];
        sprintf(result_path, "../results/%s_result.txt",
                strrchr(filename.c_str(), '/') ? strrchr(filename.c_str(), '/') + 1 : filename.c_str());
        writeVectorToFile(result_path, h_results, rows);

        float geo_avg = geometric_mean(timers, NITER);
        int num_FLOPs = 2 * h_matrix.nnz;
        int num_bytes_accessed = 5 * h_matrix.nnz * 4;
        printStats(geo_avg, num_FLOPs, num_bytes_accessed);
        saveStatsToJson(geo_avg, num_FLOPs, num_bytes_accessed, filename, "gpu_coo_add_atomic");

        // Free allocated memory
        hipFree(d_vector);
        hipFree(d_results);
        freeCOOMatrixCuda(d_matrix);
        std::free(h_vector);
        std::free(h_results);
        freeCOOMatrix(h_matrix);
    }

    return 0;
}
