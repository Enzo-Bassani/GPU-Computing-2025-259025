#include "hip/hip_runtime.h"
#include "move.h"
#include "read.h"
#include "readcu.h"
#include "utils.h"
#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#include <time.h>

// Function to multiply CSR matrix by a vector
__global__ void multiplyMatrixVector(CSRMatrix matrix, float *vector, float *result) {
    int stride = blockDim.x * gridDim.x; // = total threads
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    for (; i < matrix.numRows; i += stride) {
        float sum = 0.0;

        int start = matrix.rowPtrs[i];
        int end = matrix.rowPtrs[i + 1];

        for (int j = start; j < end; j++) {
            int col = matrix.cols[j];
            float val = matrix.values[j];
            sum += val * vector[col];
        }

        result[i] = sum;
    }
}

int main() {
    printf("Running!\n");

    // char filename[] = "../matrices/1138_bus_sorted.mtx";
    // char filename[] = "../matrices/4884_bcsstk16_sorted.mtx";
    // char filename[] = "../matrices/10974_bcsstk17_sorted.mtx";
    // char filename[] = "../matrices/36057_onetone1_sorted.mtx";
    // char filename[] = "../matrices/929901_Hardesty2_sorted.mtx";
    char filename[] = "../matrices/923136_Emilia_923_sorted.mtx";
    // Read the matrix from file
    CSRMatrix h_matrix = readMTXFileCSR(filename);
    CSRMatrix d_matrix = h_matrix;
    moveCSRToDevice(h_matrix, d_matrix);
    // printCSRMatrixHead(h_matrix);

    int cols = h_matrix.numCols, rows = h_matrix.numRows;

    // Create vector of ones
    float *h_vector = (float *)std::malloc(cols * sizeof(float));
    if (h_vector == NULL) {
        fprintf(stderr, "Memory allocation failed for vector\n");
        freeCSRMatrixCuda(h_matrix);
        exit(1);
    }

    for (int i = 0; i < cols; i++) {
        h_vector[i] = 1.0;
    }

    float *d_vector = moveArrayToDevice(h_vector, cols);

    // Multiply matrix by vector
    size_t results_size = rows * sizeof(float);
    float *h_results = (float *)std::malloc(results_size);
    float *d_results;
    hipMalloc(&d_results, results_size);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int threads_per_block = 256;
    int num_blocks = 512;
    printf("Launching %d blocks of %d threads\n", num_blocks, threads_per_block);
    hipEventRecord(start);
    multiplyMatrixVector<<<num_blocks, threads_per_block>>>(d_matrix, d_vector, d_results);
    hipEventRecord(stop);

    hipError_t cudaerr = hipDeviceSynchronize();
    if (cudaerr != hipSuccess)
        printf("kernel launch failed with error \"%s\".\n", hipGetErrorString(cudaerr));
    hipEventSynchronize(stop);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
    }
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Save results
    hipMemcpy(h_results, d_results, results_size, hipMemcpyDeviceToHost);
    char result_path[256];
    sprintf(result_path, "../results/%s_result.txt", strrchr(filename, '/') ? strrchr(filename, '/') + 1 : filename);
    writeVectorToFile(result_path, h_results, rows);

    printf("Kernel Time: %f ms\n", milliseconds);

    // Free allocated memory
    hipFree(d_vector);
    hipFree(d_results);
    freeCSRMatrixCuda(d_matrix);
    std::free(h_vector);
    std::free(h_results);
    freeCSRMatrix(h_matrix);

    return 0;
}
