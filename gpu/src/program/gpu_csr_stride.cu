#include "hip/hip_runtime.h"
#include "move.h"
#include "read.h"
#include "readcu.h"
#include "timers.h"
#include "utils.h"
#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#include <time.h>

#define WARMUP 3
#define NITER 10
// Function to multiply CSR matrix by a vector
__global__ void multiplyMatrixVector(CSRMatrix matrix, float *vector, float *result) {
    int stride = blockDim.x * gridDim.x; // = total threads
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    for (; i < matrix.numRows; i += stride) {
        float sum = 0.0;

        int start = matrix.rowPtrs[i];
        int end = matrix.rowPtrs[i + 1];

        for (int j = start; j < end; j++) {
            int col = matrix.cols[j];
            float val = matrix.values[j];
            sum += val * vector[col];
        }

        result[i] = sum;
    }
}

int main() {
    printf("Running!\n");

    // char filename[] = "../matrices/1138_bus_sorted.mtx";
    // char filename[] = "../matrices/4884_bcsstk16_sorted.mtx";
    // char filename[] = "../matrices/10974_bcsstk17_sorted.mtx";
    // char filename[] = "../matrices/36057_onetone1_sorted.mtx";
    // char filename[] = "../matrices/929901_Hardesty2_sorted.mtx";
    // char filename[] = "../matrices/923136_Emilia_923_sorted.mtx";
    // Read the matrix from file
    auto filenames = getFilenames();
    for (std::string filename : filenames) {
        CSRMatrix h_matrix = readMTXFileCSR(filename.c_str());
        CSRMatrix d_matrix = h_matrix;
        moveCSRToDevice(h_matrix, d_matrix);
        // printCSRMatrixHead(h_matrix);

        int cols = h_matrix.numCols, rows = h_matrix.numRows;

        // Create vector of ones
        float *h_vector = (float *)std::malloc(cols * sizeof(float));
        if (h_vector == NULL) {
            fprintf(stderr, "Memory allocation failed for vector\n");
            freeCSRMatrixCuda(h_matrix);
            exit(1);
        }

        for (int i = 0; i < cols; i++) {
            h_vector[i] = 1.0;
        }

        float *d_vector = moveArrayToDevice(h_vector, cols);

        // Multiply matrix by vector
        size_t results_size = rows * sizeof(float);
        float *h_results = (float *)std::malloc(results_size);
        float *d_results;
        hipMalloc(&d_results, results_size);

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        int threads_per_block = 256;
        int num_blocks = 512;
        printf("Launching %d blocks of %d threads\n", num_blocks, threads_per_block);
        float timers[NITER];
        float iter_time = 0;
        for (int i = -WARMUP; i < NITER; i++) {
            hipMemset(d_results, 0, results_size);
            hipEventRecord(start);
            multiplyMatrixVector<<<num_blocks, threads_per_block>>>(d_matrix, d_vector, d_results);
            hipEventRecord(stop);

            hipError_t cudaerr = hipDeviceSynchronize();
            if (cudaerr != hipSuccess)
                printf("kernel launch failed with error \"%s\".\n", hipGetErrorString(cudaerr));

            hipEventSynchronize(stop);
            hipEventElapsedTime(&iter_time, start, stop);
            if (i >= 0) {
                timers[i] = iter_time;
                printf("%d iter_time %f\n", i, iter_time);
            }
        }

        // Save results
        hipMemcpy(h_results, d_results, results_size, hipMemcpyDeviceToHost);
        char result_path[256];
        sprintf(result_path, "../results/%s_result.txt",
                strrchr(filename.c_str(), '/') ? strrchr(filename.c_str(), '/') + 1 : filename.c_str());
        writeVectorToFile(result_path, h_results, rows);

        float geo_avg = geometric_mean(timers, NITER);
        int num_FLOPs = 2 * h_matrix.nnz;
        int num_bytes_accessed = (3 * rows + 3 * h_matrix.nnz) * 4;
        printStats(geo_avg, num_FLOPs, num_bytes_accessed);
        saveStatsToJson(geo_avg, num_FLOPs, num_bytes_accessed, filename, "gpu_csr_stride");

        // Free allocated memory
        hipFree(d_vector);
        hipFree(d_results);
        freeCSRMatrixCuda(d_matrix);
        std::free(h_vector);
        std::free(h_results);
        freeCSRMatrix(h_matrix);
    }
    return 0;
}
