#include "read.h"
#include "readcu.h"
#include <stdlib.h>

// Function to free COO matrix memory
void freeCOOMatrixCuda(COOMatrix matrix) {
    hipFree(matrix.rows);
    hipFree(matrix.cols);
    hipFree(matrix.values);
}

// Function to free CSR matrix memory
void freeCSRMatrixCuda(CSRMatrix matrix) {
    hipFree(matrix.rowPtrs); // Free the row pointers array
    hipFree(matrix.cols);    // Free the column indices array
    hipFree(matrix.values);  // Free the values array
}
