#include "move.h"
#include "read.h"
#include <cstdio>

void moveCOOToDevice(COOMatrix &h, COOMatrix &d) {
    size_t rowsSize = (d.nnz * sizeof(int));
    size_t colsSize = (d.nnz * sizeof(int));
    size_t valuesSize = (d.nnz * sizeof(float));

    checkCudaError(hipMalloc(&d.rows, rowsSize));
    checkCudaError(hipMalloc(&d.cols, colsSize));
    checkCudaError(hipMalloc(&d.values, valuesSize));

    checkCudaError(hipMemcpy(d.rows, h.rows, rowsSize, hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(d.cols, h.cols, colsSize, hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(d.values, h.values, valuesSize, hipMemcpyHostToDevice));
}

void moveCSRToDevice(CSRMatrix &h, CSRMatrix &d) {
    size_t rowPtrsSize = ((d.numRows + 1) * sizeof(int));
    size_t colsSize = (d.nnz * sizeof(int));
    size_t valuesSize = (d.nnz * sizeof(float));

    checkCudaError(hipMalloc(&d.rowPtrs, rowPtrsSize));
    checkCudaError(hipMalloc(&d.cols, colsSize));
    checkCudaError(hipMalloc(&d.values, valuesSize));

    checkCudaError(hipMemcpy(d.rowPtrs, h.rowPtrs, rowPtrsSize, hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(d.cols, h.cols, colsSize, hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(d.values, h.values, valuesSize, hipMemcpyHostToDevice));
}

void checkCudaError(hipError_t err) {
    if (err != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
        exit(1);
    }
}

float *moveArrayToDevice(float *h_array, int length) {
    float *d_array;
    size_t size = sizeof(float) * length;
    checkCudaError(hipMalloc(&d_array, size));
    hipMemcpy(d_array, h_array, size, hipMemcpyHostToDevice);
    return d_array;
}
